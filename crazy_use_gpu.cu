
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

const long long tdelay = 1000000LL;
const int hdelay = 1000;

__global__ void dkern(){

  long long start = clock64();
  while(clock64() < start + tdelay);
}

int main(int argc, char *argv[]){
  int i = 0;
  int my_delay = hdelay;
  if (argc > 1) my_delay = atoi(argv[1]);
  while (i == 0){
    dkern<<<1, 1>>>();
    usleep(my_delay);
  }
  return 0;
}
